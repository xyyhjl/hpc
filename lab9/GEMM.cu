#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
int main() {
    // 设置矩阵的维度
    int m = 2048;
    int n = 2048;
    int k = 23048; 

    // 分配并初始化CPU上的输入矩阵
    float* A = (float*)malloc(m * k * sizeof(float));
    float* B = (float*)malloc(k * n * sizeof(float));
    for (int i = 0; i < m * k; i++) {
        A[i] = i;
    }
    for (int i = 0; i < k * n; i++) {
        B[i] = i;
    }

    // 在GPU上分配内存
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

    // 创建cuBLAS句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 创建CUDA事件来测量时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // 执行矩阵乘法
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    // 停止计时器
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 输出运行时间
    printf("GPU:Matrix multiplication took %f milliseconds.\n", milliseconds);

    // 将结果从设备复制到主机
    float* C = (float*)malloc(m * n * sizeof(float));
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    //实现朴素矩阵乘法
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);
    for(int i=0;i<m;i++)
    {
    	for(int j=0;j<n;j++)
    	{
    		for(int p=0;p<k;p++)
    		{
    			C[i*n+j]+=A[i*k+p]*B[p*n+j];
    		}

    	}
    }
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    printf("Naive:Matrix multiplication took %f milliseconds.\n", milliseconds2);
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    return 0;
}