#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>

#define m 2048;
#define n 2048;
#define k 2048;
int main() {
    float* A = (float*)malloc(m * k * sizeof(float));
    float* B = (float*)malloc(k * n * sizeof(float));
    for (int i = 0; i < m * k; i++) {
        A[i] = i;
    }
    for (int i = 0; i < k * n; i++) {
        B[i] = i;
    }
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU:Matrix multiplication took %f milliseconds.\n", milliseconds);
    float* C = (float*)malloc(m * n * sizeof(float));
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);
    for(int i=0;i<m;i++)
    {
    	for(int j=0;j<n;j++)
    	{
    		for(int p=0;p<k;p++)
    		{
    			C[i*n+j]+=A[i*k+p]*B[p*n+j];
    		}

    	}
    }
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    printf("Naive:Matrix multiplication took %f milliseconds.\n", milliseconds2);
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    return 0;
}